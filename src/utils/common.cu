#include "common.hpp"
#include "console.hpp"

fn gpu_assert(hipError_t code, const char *file, int line) -> void {
  if (code != hipSuccess) {
    console::error("GPU: ", hipGetErrorString(code), " in ", file, ":", line);
    exit(code);
  }
}
